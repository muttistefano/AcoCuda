#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <fstream>
#include <iostream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <unistd.h>
#include <ctime>
#include <cstdlib>
#define Infinity 65536
#define randomize() srand((unsigned)time(NULL))
#define index(length,line,column) (column + line * length)


struct joints{
    float joint1;
    float joint2;
    float joint3;
    float joint4;
    float joint5;
    float joint6;
    bool ch = false;
    float ph = 0.0;
};


//////////DEVICE FUNCTIONS

/*__device__ double atomicMul(double* address, double val) 
{ 
 unsigned long long int* address_as_ull = (unsigned long long int*)address; 
 unsigned long long int old = *address_as_ull, assumed; 
 do { 
 assumed = old; 
 old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val * __longlong_as_double(assumed))); 
 } while (assumed != old);
 return __longlong_as_double(old);
}     */ 

__device__ float atomicMul(float* address, float val) 
{
  int* address_as_int = (int*)address; 
  int old = *address_as_int, assumed; 
  do { 
    assumed = old; 
    old = atomicCAS(address_as_int, assumed, __float_as_int(val * 
__float_as_int(assumed))); 
 } while (assumed != old); return __int_as_float(old);
}

__global__ void Cycle(int n_pnt,int n_conf,int n_ants,joints* dev_graph_ptr,unsigned int seed)
{
  hiprandState_t state;

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int index_ch = index * n_pnt;
  float rnd_sel,prev_ph,tot_ph;

  __shared__ int sol[10000]; //controlla lunghezza o dynamic 
//   int* sol = new int[n_pnt];
  
  hiprand_init(clock(),index,0, &state);
  
  for (int pnt=0 ; pnt<n_pnt ; pnt++) //PROBABILISTIC SELECTION IMPLEMENTATION
  {
    prev_ph=0;
    tot_ph =0;
    rnd_sel=hiprand_uniform(&state);
    
    for(int cht=0;cht<n_conf;cht++)
    {
      tot_ph=tot_ph+(*(dev_graph_ptr+pnt+cht*n_pnt)).ph; //SHARED MEMORY <----
    }
    
    rnd_sel = rnd_sel * tot_ph;
//     printf("randomnum:%f  \n",rnd_sel);
    
    for(int conf=0;conf<n_conf;conf++)
    {
      prev_ph=prev_ph+(*(dev_graph_ptr+pnt+conf*n_pnt)).ph;
      if(rnd_sel<prev_ph)
      {
	sol[threadIdx.x*n_pnt + pnt]=conf;
	break;
      }
    }

  }
  /*
  for(int gg=0;gg<n_pnt;gg++){
    printf(" %d ",sol[threadIdx.x*n_pnt + gg]);
  }*/
  printf(" %d ",sol[threadIdx.x*n_pnt]);
//   printf("\n ");
  
  __syncthreads();
  
  for(int q=0;q<n_ants;q++) //PH VALUE ADDING ---- n_threads α n_points ---- OPTIMIZE 
  {
    atomicAdd(&(*(dev_graph_ptr+threadIdx.x+n_pnt*sol[q*n_pnt+threadIdx.x])).ph,0.1); //BOH
//     atomicMul(&(*(dev_graph_ptr+threadIdx.x)).ph,1.02);
  }
  
  for(int mm=0;mm<(int)((n_conf*n_pnt)/n_ants);mm++)  //FIX THIS FOR EVERY CASE 
  {
    if((*(dev_graph_ptr+threadIdx.x+n_pnt*mm)).ph > 0.2 & (*(dev_graph_ptr+threadIdx.x+n_pnt*mm)).ch)
    {
      atomicAdd(&(*(dev_graph_ptr+threadIdx.x+n_pnt*mm)).ph,-0.1);
    }
  }
}

__global__ void print_matrix(joints* ptr,int n_points,int n_conf){
    for(int i=0; i < n_points*n_conf; ++i){
        printf("%f ",ptr->ph);
	if (i%n_points==(n_points-1)) printf("\n");
        ptr++;
    }
}


///////////CLASS


class AcoCuda
{
    int n_points;
    int n_conf;
    int n_ants;

    thrust::host_vector<joints>   host_graph;
    thrust::host_vector<int>      host_path;
    thrust::device_vector<joints> device_graph;
    joints*                       device_graph_ptr;

    
  public:

    AcoCuda(int n_points,int n_conf,int n_ants);
    
    void LoadGraph();
    void PhInit();
    void Phrenew();
    void Phevaporate();
    
    void RunCycle();
    void RunPrint();

};

///////////CLASS METHODS

AcoCuda::AcoCuda(int n_pointsex, int n_confex, int n_antsex)
{
  n_ants=n_antsex;
  n_conf=n_confex;
  n_points=n_pointsex;
  thrust::host_vector<joints> tmp(n_pointsex*n_confex);
  host_graph=tmp;
}

void AcoCuda::LoadGraph()
{
  srand(time(NULL));
  printf("points: %d\n",n_points);
  printf("config: %d\n",n_conf);

  for(thrust::host_vector<joints>::iterator j = host_graph.begin(); j != host_graph.end(); j++){
    (*j).joint1=rand()/(RAND_MAX/3);
    (*j).joint2=rand()/(RAND_MAX/3);
    (*j).joint3=rand()/(RAND_MAX/3);
    (*j).joint4=rand()/(RAND_MAX/3);
    (*j).joint5=rand()/(RAND_MAX/3);
    (*j).joint6=rand()/(RAND_MAX/3);

 if(rand()<(RAND_MAX*0.6)){
      (*j).ch=true;
    }      
  
  }

}

void AcoCuda::PhInit()
{
  float n_act;
  int ind;
  ind=0;
  std::vector<float> ph_ind;
  ph_ind.clear();
  for(thrust::host_vector<joints>::iterator j = host_graph.begin(); j != host_graph.begin()+n_points; j++){
    n_act=0;
    for (int u=0;u<n_conf;u++)
    {
      n_act=n_act+(*(j+u*n_points)).ch;
    }
//     printf("%d\n",n_act);
    n_act = 1/n_act;
    ph_ind.push_back(n_act);
  }
  for(thrust::host_vector<joints>::iterator z = host_graph.begin(); z != host_graph.begin()+n_points; z++){
    
    for (int uu=0;uu<n_conf;uu++)
    {
      if ((*(z+uu*n_points)).ch){
	 (*(z+uu*n_points)).ph=ph_ind[ind];
      }
      else{
	(*(z+uu*n_points)).ph=0;
      }
    }
    ind++;
  }

   device_graph=host_graph;
   device_graph_ptr = thrust::raw_pointer_cast((device_graph.data()));
//    device_graph_ptr = thrust::raw_pointer_cast(&device_graph[0]);
}


/////////////METHODS FOR CALLING DEVICES FUNCTIONS

void AcoCuda::RunCycle()
{
  Cycle<<<1,n_ants >>>(n_points,n_conf,n_ants,device_graph_ptr,time(NULL));//<<<blocks,thread>>>
  
}

void AcoCuda::RunPrint()
{
  print_matrix<<< 1,1 >>>(this->device_graph_ptr,n_points,n_conf);
}

////////////MAIN

int main(){
 
  AcoCuda test(10,8,10);//points,conf,ants ---- MAX 128 pnt con 8 configurazioni per ora

  test.LoadGraph();
  test.PhInit();
  
  test.RunPrint();
  hipDeviceSynchronize();
  
  for (int y=0;y<1;y++)
  {
    test.RunCycle();
    hipDeviceSynchronize();
  }
  printf("\n");
  test.RunPrint();
  hipDeviceSynchronize();
  
  printf("\nEnd\n");
  //test.print_matrix();
  
  return 0;
}


