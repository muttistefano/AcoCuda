#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <fstream>
#include <iostream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <unistd.h>
#include <ctime>
#include <cstdlib>
#define Infinity 65536
#define randdouble() ((double)rand()/(double)RAND_MAX)
#define randomize() srand((unsigned)time(NULL))
#define index(length,line,column) (column + line * length)


struct joints{
    double joint1;
    double joint2;
    double joint3;
    double joint4;
    double joint5;
    double joint6;
    bool ch = false;
    double ph = 0.0;
};


//////////DEVICE FUNCTIONS


__global__ void Cycle(int n_pnt,int n_conf,joints* dev_graph_ptr,unsigned int seed)
{
  hiprandState_t state;

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int index_ch = index * n_pnt;
  float rnd_sel,prev_ph;
  int choice=0;
  
  hiprand_init(clock(),index,0, &state);
  
  //printf("index:%d\n",index);
  
  
  for (int pnt=0 ; pnt<n_pnt ; pnt++)
  {
    prev_ph=0;
    choice=0;
    rnd_sel=hiprand_uniform(&state);
    printf("randomnum:%f  ",rnd_sel);
    for(int conf=0;conf<n_conf;conf++)
    {
      prev_ph=prev_ph+(*(dev_graph_ptr+pnt+conf*n_pnt)).ph;
      if(rnd_sel<prev_ph)
      {
	choice=conf;
	printf("%d \n",choice);
	break;
      }
      
    }

  }
  
}

__global__ void print_matrix(joints* ptr,int n_points,int n_conf){
    for(int i=0; i < n_points*n_conf; ++i){
        printf("%f ",ptr->ph);
	if (i%n_points==(n_points-1)) printf("\n");
        ptr++;
    }
}

///////////CLASS


class AcoCuda
{
    int n_points;
    int n_conf;
    int n_ants;
    
    thrust::host_vector<joints>   host_graph;
    thrust::host_vector<int>      host_path;
    thrust::device_vector<joints> device_graph;
    joints*                       device_graph_ptr;
//     thrust::host_vector<double>   host_ph;
//     thrust::device_vector<double> device_ph;
//     double*                       device_ph_ptr;
    
  public:

    AcoCuda(int n_points,int n_conf,int n_ants);
    
    void LoadGraph();
    void PhInit();
    void Phrenew();
    void Phevaporate();
    
    void RunCycle();
    void RunPrint();

};

///////////CLASS METHODS

AcoCuda::AcoCuda(int n_pointsex, int n_confex, int n_antsex)
{
  n_ants=n_antsex;
  n_conf=n_confex;
  n_points=n_pointsex;
  thrust::host_vector<joints> tmp(n_pointsex*n_confex);
  host_graph=tmp;
}

void AcoCuda::LoadGraph()
{
  printf("points: %d\n",n_points);
  printf("config: %d\n",n_conf);

  for(thrust::host_vector<joints>::iterator j = host_graph.begin(); j != host_graph.end(); j++){
    (*j).joint1=(double)rand()/(RAND_MAX/3);
    (*j).joint2=(double)rand()/(RAND_MAX/3);
    (*j).joint3=(double)rand()/(RAND_MAX/3);
    (*j).joint4=(double)rand()/(RAND_MAX/3);
    (*j).joint5=(double)rand()/(RAND_MAX/3);
    (*j).joint6=(double)rand()/(RAND_MAX/3);

 if(rand()<(RAND_MAX*0.8)){
      (*j).ch=true;
    }      
  
  }

}

void AcoCuda::PhInit()
{
  float n_act;
  int ind;
  ind=0;
  std::vector<double> ph_ind;
  ph_ind.clear();
  for(thrust::host_vector<joints>::iterator j = host_graph.begin(); j != host_graph.begin()+n_points; j++){
    n_act=0;
    for (int u=0;u<n_conf;u++)
    {
      n_act=n_act+(*(j+u*n_points)).ch;
    }
//     printf("%d\n",n_act);
    n_act = 1/n_act;
    ph_ind.push_back(n_act);
  }
  for(thrust::host_vector<joints>::iterator z = host_graph.begin(); z != host_graph.begin()+n_points; z++){
    
    for (int uu=0;uu<n_conf;uu++)
    {
      if ((*(z+uu*n_points)).ch){
	 (*(z+uu*n_points)).ph=ph_ind[ind];
      }
      else{
	(*(z+uu*n_points)).ph=0;
      }
    }
    ind++;
  }

   device_graph=host_graph;
   device_graph_ptr = thrust::raw_pointer_cast((device_graph.data()));
//    device_graph_ptr = thrust::raw_pointer_cast(&device_graph[0]);
}


/////////////METHODS FOR CALLING DEVICES FUNCTIONS

void AcoCuda::RunCycle()
{
  Cycle<<< 1,1 >>>(n_points,n_conf,device_graph_ptr,time(NULL));//<<<blocks,thread>>>
  hipDeviceSynchronize();
}

void AcoCuda::RunPrint()
{
  print_matrix<<< 1,1 >>>(this->device_graph_ptr,n_points,n_conf);
//   hipDeviceSynchronize();
}

////////////MAIN

int main(){
  int viao=0;
  

  AcoCuda test(18,6,1000);//points,conf,ants

  test.LoadGraph();
  test.PhInit();
  
  test.RunPrint();

  test.RunCycle();

  //test.print_matrix();
  
  return 0;
}


